#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "NewtonMath.h"
#include "JuliaMath.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void newton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void newton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n)
    {
    float epsilon = 0.1f;
    NewtonMath newtonMath = NewtonMath(epsilon, n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH = w * h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

	newtonMath.colorXY(&color, x, y); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}

    // must be present !
    //delete newtonMath;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
