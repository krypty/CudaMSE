#include "hip/hip_runtime.h"
#include <assert.h>

#include "Newton.h"
#include "Device.h"
#include "MathTools.h"

using cpu::IntervalI;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void newton(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Newton::Newton(int w, int h, int nMin, int nMax) :
	variateurN(IntervalI(nMin, nMax), 1)
    {
    // Inputs
    this->w = w;
    this->h = h;

    this->ptrDomaineMathInit = new DomaineMath(-1.3, -1.4, 1.4, 1.3);

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser

    //Outputs
    this->title = "Newton_CUDA (Zoomable)";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    }

Newton::~Newton()
    {
    delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Newton::process(uchar4* ptrDevPixels, int w, int h, const DomaineMath& domaineMath)
    {
newton<<<dg,db>>>(ptrDevPixels,w,h,domaineMath, n);
}

/**
 * Override
 * Call periodicly by the API
 */
void Newton::animationStep()
{
this->n = variateurN.varierAndGet(); // in [0,2pi]
}

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* Newton::getDomaineMathInit(void)
{
return ptrDomaineMathInit;
}

/**
 * Override
 */
float Newton::getAnimationPara(void)
{
return n;
}

/**
 * Override
 */
int Newton::getW(void)
{
return w;
}

/**
 * Override
 */
int Newton::getH(void)
{
return h;
}

/**
 * Override
 */
string Newton::getTitle(void)
{
return title;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
