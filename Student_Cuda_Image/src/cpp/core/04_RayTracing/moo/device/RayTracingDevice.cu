#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"
#include "RayTracingMath.h"
#include "Sphere.h"

// on fait des define car conditions ne bougent plus à l'exécution et c'est plus performant que les IF qui doivent être évalués à chaque itération
// Il ne peut y avoir qu'une seule valeur a true possible
#define USE_GM false
#define USE_CM false
#define USE_SM true

/*----------------------------------------------------------------------*\
|*			Declaration                     *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Imported	    *|
\*-------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, int w, int h, Sphere* ptrDevSpheres, int n, float t);

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

static __device__ void copyGMtoSM(Sphere* tabSpheresSM, Sphere* ptrDevSpheresGM, int n);

static __device__ void work(int w, int h, int n, float t, Sphere* ptrSpheres, uchar4* ptrDevPixels);

/*----------------------------------------------------------------------*\
|*			Implementation                  *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

__global__
void rayTracing(uchar4* ptrDevPixels, int w, int h, Sphere* ptrDevSpheresGM, int n, float t)
{
 #if USE_SM
        extern __shared__ Sphere tabSpheresSM[];
        copyGMtoSM(tabSpheresSM, ptrDevSpheresGM, n);
        __syncthreads();
        work(w, h, n, t, tabSpheresSM, ptrDevPixels);
 #else
        work(w, h, n, t, ptrDevSpheresGM, ptrDevPixels);
 #endif

}

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

__device__
void work(int w, int h, int n, float t, Sphere* ptrDevSpheres, uchar4* ptrDevPixels)
{
        RayTracingMath rayTracingMath = RayTracingMath(ptrDevSpheres, n, t);
        const int WH = w * h;
        const int NB_THREAD = Indice2D::nbThread();
        const int TID = Indice2D::tid();
        int s = TID;
        int i;
        int j;
        uchar4 color;
        while (s < WH)
        {
                IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[
                // rayTracingMath.color(i, j, t, ptrDevPixels[s]);
                rayTracingMath.colorIJ(&color,i, j);  // update color
                ptrDevPixels[s] = color;
                s += NB_THREAD;
        }
}

__device__
void copyGMtoSM(Sphere* tabSpheresSM, Sphere* ptrDevSpheresGM, int n)
{
        const int TID_LOCAL = Indice2D::tidLocal();
        const int NB_THREAD_BLOCK = Indice2D::nbThreadBlock();

        int s = TID_LOCAL;

        while (s < n)
        {
                tabSpheresSM[s] = ptrDevSpheresGM[s];
                s += NB_THREAD_BLOCK;
        }
}

/*----------------------------------------------------------------------*\
|*			End	                    *|
\*---------------------------------------------------------------------*/
