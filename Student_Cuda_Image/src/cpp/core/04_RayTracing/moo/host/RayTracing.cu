#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "RayTracing.h"
#include "Device.h"
#include "MathTools.h"
#include "SphereFactory.h"
#include "ConstantMemoryLink.h"

using cpu::IntervalI;

/* on fait des define car conditions ne bougent plus à l'exécution et c'est plus performant que les IF qui doivent être évalués à chaque itération
 * Ces valeurs doivent être cohérente avec celles présentes dans RayTracingDevice.cu
*/
//TODO PROF: jouer avec ces valeurs. Attention: Il doit y avoir une seule et unique valeur a true.
#define USE_GM false
#define USE_CM false
#define USE_SM true

#define LENGTH 1000
__constant__ Sphere TAB_DATA_CM[LENGTH];

/*----------------------------------------------------------------------*\
|*			Declaration                     *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Imported	    *|
\*-------------------------------------*/

extern __global__ void rayTracing(uchar4* ptrDevPixels, int w, int h, Sphere* ptrDevSpheres, int n, float t);

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

/*----------------------------------------------------------------------*\
|*			Implementation                  *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

/*-------------------------*\
|*	Constructeur	    *|
\*-------------------------*/

RayTracing::RayTracing(int w, int h, int nSphere, float dt)
{
        // Inputs
        this->w = w;
        this->h = h;
        this->nSphere = nSphere;
        this->dt = dt;

        // Tools
        this->dg = dim3(16, 2, 1); // disons a optimiser
        this->db = dim3(32, 4, 1); // disons a optimiser

        int margin = 50;
        this->ptrSpheres = SphereFactory::createSpheres(nSphere, w, h, margin);

        this->t = 0.0f;

        //Outputs
        this->title = "RayTracing_CUDA";

        // Check:
        //print(dg, db);
        Device::assertDim(dg, db);

    #if USE_CM
        copySpheresToConstantMemory();
    #endif

        sizeSpheres = sizeof(Sphere) * LENGTH;

    #if USE_GM == true || USE_SM == true
        HANDLE_ERROR(hipMalloc(&ptrDevSpheres, sizeSpheres));
        HANDLE_ERROR(hipMemcpy(ptrDevSpheres, ptrSpheres, sizeSpheres, hipMemcpyHostToDevice));
    #endif
}

RayTracing::~RayTracing()
{
        delete[] this->ptrSpheres;
        HANDLE_ERROR(hipFree(ptrDevSpheres));
}

/*-------------------------*\
|*	Methode		    *|
\*-------------------------*/

ConstantMemoryLink constantMemoryLink(void)
{
        Sphere* ptrDevTabData;
        size_t sizeAll = LENGTH * sizeof(Sphere);
        HANDLE_ERROR(hipGetSymbolAddress((void ** )&ptrDevTabData, TAB_DATA_CM));
        ConstantMemoryLink cmLink =
        {
                (void**) ptrDevTabData, LENGTH, sizeAll
        };
        return cmLink;
}

void RayTracing::copySpheresToConstantMemory()
{
        ConstantMemoryLink cmLink = constantMemoryLink();
        this->ptrDevSpheres = (Sphere*) cmLink.ptrDevTab;
        size_t sizeALL = cmLink.sizeAll;

        HANDLE_ERROR(hipMemcpy(ptrDevSpheres, ptrSpheres, sizeALL, hipMemcpyHostToDevice));
}

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::process(uchar4* ptrDevPixels, int w, int h)
{
        // rayTracing<<<dg,db>>>(ptrDevPixels, w, h, ptrDevSpheres, this->nSphere, t);

        #if USE_GM == true || USE_CM == true
        rayTracing<<<dg, db>>>(ptrDevPixels, w, h, ptrDevSpheres, this->nSphere, t);
        #endif

        #if USE_SM
        rayTracing<<<dg, db, sizeSpheres>>>(ptrDevPixels, w, h, ptrDevSpheres, this->nSphere, t);
        #endif
}

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
{
        t += dt;
}

/*--------------*\
|*	get	 *|
\*--------------*/

/**
 * Override
 */
float RayTracing::getAnimationPara(void)
{
        return t;
}

/**
 * Override
 */
int RayTracing::getW(void)
{
        return w;
}

/**
 * Override
 */
int RayTracing::getH(void)
{
        return h;
}

/**
 * Override
 */
string RayTracing::getTitle(void)
{
        return title;
}

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

/*----------------------------------------------------------------------*\
|*			End	                    *|
\*---------------------------------------------------------------------*/
