#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "MandelbrotMath.h"
#include "JuliaMath.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n)
    {
    // hiérarchie de classe et polymorphisme (donc en utilisant des pointeurs) fonctionnels

    float c1 = -0.12;
    float c2 = 0.85;

    //TODO PROF: commenter ou décommenter pour passer de Mandelbrot à Julia ou vice versa
    //TODO PROF: voir version OpenMP pour tous les autres représentations de fractales
//    MandelbrotMathBase* mandelbrotMath = new JuliaMath(n, c1, c2);
    MandelbrotMathBase* mandelbrotMath = new MandelbrotMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH = w * h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

	mandelbrotMath->colorXY(&color, x, y); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}

    // must be present !
    delete mandelbrotMath;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
