#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "DomaineMath.h"
#include "cudaTools.h"
#include "Device.h"
#include "MandelbrotMath.h"
#include "JuliaMath.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n)
    {
    // hiérarchie de classe fonctionnelle
    // mais polymorphisme (donc en utilisant des pointeurs) non fonctionnel

    //MandelbrotMath mandelbrotMath = MandelbrotMath(n);

    float c1 = -0.12;
    float c2 = 0.85;
    JuliaMath mandelbrotMath = JuliaMath(n, c1, c2);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    const int WH = w * h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

	mandelbrotMath.colorXY(&color, x, y); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}

    // must be present !
    //delete mandelbrotMath;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
