#include "hip/hip_runtime.h"
#include <assert.h>

#include "Mandelbrot.h"
#include "Device.h"
#include "MathTools.h"

using cpu::IntervalI;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Mandelbrot::Mandelbrot(int w, int h, int nMin, int nMax) :
	variateurN(IntervalI(nMin, nMax), 1)
    {
    // Inputs
    this->w = w;
    this->h = h;

    this->ptrDomaineMathInit = new DomaineMath(-2.1, -1.3, 0.8, 1.3); // Mandelbrot
//    this->ptrDomaineMathInit = new DomaineMath(-1.3, -1.4, 1.4, 1.3); // Julia

    // Tools
    this->dg = dim3(8, 8, 1); // disons a optimiser
    this->db = dim3(16, 16, 1); // disons a optimiser

    //Outputs
    this->title = "Mandelbrot_CUDA (Zoomable)";

    // Check:
    //print(dg, db);
    Device::assertDim(dg, db);
    }

Mandelbrot::~Mandelbrot()
    {
    delete ptrDomaineMathInit;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::process(uchar4* ptrDevPixels, int w, int h, const DomaineMath& domaineMath)
    {
mandelbrot<<<dg,db>>>(ptrDevPixels,w,h,domaineMath, n);
}

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::animationStep()
{
this->n = variateurN.varierAndGet(); // in [0,2pi]
}

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
DomaineMath* Mandelbrot::getDomaineMathInit(void)
{
return ptrDomaineMathInit;
}

/**
 * Override
 */
float Mandelbrot::getAnimationPara(void)
{
return n;
}

/**
 * Override
 */
int Mandelbrot::getW(void)
{
return w;
}

/**
 * Override
 */
int Mandelbrot::getH(void)
{
return h;
}

/**
 * Override
 */
string Mandelbrot::getTitle(void)
{
return title;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

