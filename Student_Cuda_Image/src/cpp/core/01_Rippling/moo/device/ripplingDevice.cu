#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools.h"

#include "RipplingMath.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t)
    {
    RipplingMath ripplingMath = RipplingMath(w, h);

    // TODO pattern entrelacement

    const int NB_THREAD = Indice2D::nbThread();

    int TID = Indice2D::tid();
    int s = TID;
    const int WH = w*h;

    while(s < WH)
	{
	int i = 0;
	int j = 0;
	uchar4 color;

	IndiceTools::toIJ(s, w, &i, &j);

	ripplingMath.colorIJ(i, j, t, &color);
	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

