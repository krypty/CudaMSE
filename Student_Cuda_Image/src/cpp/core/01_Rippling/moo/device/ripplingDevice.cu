#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools.h"

#include "RipplingMath.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration                     *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Imported	    *|
\*-------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

/*----------------------------------------------------------------------*\
|*			Implementation                  *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t)
{
        RipplingMath ripplingMath = RipplingMath(w, h);

        // TODO pattern entrelacement

        // Indice2D
        const int NB_THREAD = Indice2D::nbThread();
        int TID = Indice2D::tid();

//// Indice1D
//    const int NB_THREAD = Indice1D::nbThread();
//    int TID = Indice1D::tid();

//// 1-1
//    const int NB_THREAD = (64*16) * (64*16);

        int s = TID;
        const int WH = w*h;

        while(s < WH)
        {
                int i = 0;
                int j = 0;
                uchar4 color;

                IndiceTools::toIJ(s, w, &i, &j);

                ripplingMath.colorIJ(i, j, t, &color);
                ptrDevPixels[s] = color;

                s += NB_THREAD;
        }


//	uchar4 color;
//	int i = threadIdx.y  + blockIdx.y * blockDim.y;
//	int j = threadIdx.x  + blockIdx.x * blockDim.x;
//
//	int s = i * w + j;
//	ripplingMath.colorIJ(i, j, t, &color);
//	ptrDevPixels[s] = color;
}


/*----------------------------------------------------------------------*\
|*			End	                    *|
\*---------------------------------------------------------------------*/
