#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Rippling.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration                     *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Imported	    *|
\*-------------------------------------*/

extern __global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

/*----------------------------------------------------------------------*\
|*			Implementation                  *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

/*-------------------------*\
|*	Constructeur	    *|
\*-------------------------*/

Rippling::Rippling(int w, int h, float dt, dim3 dg, dim3 db)
{
        assert(w == h);

        // Inputs
        this->w = w;
        this->h = h;
        this->dt = dt;

        // Tools

        // Indice2D
        // this->dg = dim3(64*4, 4, 1);
        // this->db = dim3(32*4, 8, 1);
        // this->dg = dim3(96, 4, 1);
        // this->db = dim3(128, 4, 1);
        this->dg = dg;
        this->db = db;


        // Indice1D
//    this->dg = dim3(512, 1, 1);
//    this->db = dim3(1024, 1, 1);

//    // pattern 1-1
//    this->dg = dim3(64, 64, 1);
//    this->db = dim3(16, 16, 1);

        this->t = 0;

        // Outputs
        this->title = "Rippling_Cuda";

        //print(dg, db);
        Device::assertDim(dg, db);
}

Rippling::Rippling(int w, int h, float dt):Rippling(w, h, dt, dim3(96, 4, 1), dim3(128, 4, 1)) {}

Rippling::~Rippling()
{
        // rien
}

/*-------------------------*\
|*	Methode		    *|
\*-------------------------*/


/**
 * Override
 */
void Rippling::process(uchar4* ptrDevPixels, int w, int h)
{
        rippling<<<dg, db>>>(ptrDevPixels, w, h, t);
}


/**
 * Override
 */
void Rippling::animationStep()
{
        this->t += this->dt;
}

/*--------------*\
|*	get	 *|
\*--------------*/

/**
 * Override
 */
float Rippling::getAnimationPara(void)
{
        return t;
}

/**
 * Override
 */
int Rippling::getW(void)
{
        return w;
}

/**
 * Override
 */
int Rippling::getH(void)
{
        return h;
}

/**
 * Override
 */
string Rippling::getTitle(void)
{
        return title;
}


/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

/*----------------------------------------------------------------------*\
|*			End	                    *|
\*---------------------------------------------------------------------*/
