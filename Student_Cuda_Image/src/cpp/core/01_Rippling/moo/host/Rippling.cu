#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Rippling.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Rippling::Rippling(int w, int h, float dt)
    {
    assert(w == h);

    // Inputs
    this->w = w;
    this->h = h;
    this->dt = dt;

    // Tools

//    // Indice2D
    this->dg = dim3(64*4, 4, 1);
    this->db = dim3(32*4, 8, 1);

    // Indice1D
//    this->dg = dim3(512, 1, 1);
//    this->db = dim3(1024, 1, 1);

//    // pattern 1-1
//    this->dg = dim3(64, 64, 1);
//    this->db = dim3(16, 16, 1);

    this->t = 0;

    // Outputs
    this->title = "Rippling_Cuda";

    //print(dg, db);
    Device::assertDim(dg, db);
    }

Rippling::~Rippling()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/


/**
 * Override
 */
void Rippling::process(uchar4* ptrDevPixels, int w, int h)
    {
    rippling<<<dg, db>>>(ptrDevPixels, w, h, t);
    }


/**
 * Override
 */
void Rippling::animationStep()
    {
    this->t += this->dt;
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float Rippling::getAnimationPara(void)
    {
    return t;
    }

/**
 * Override
 */
int Rippling::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Rippling::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string Rippling::getTitle(void)
    {
    return title;
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

