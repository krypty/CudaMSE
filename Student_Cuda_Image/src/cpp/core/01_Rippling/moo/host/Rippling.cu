#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Rippling.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rippling(uchar4* ptrDevPixels, int w, int h, float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Rippling::Rippling(int w, int h, float dt)
    {
    assert(w == h);

    // Inputs
    this->w = w;
    this->h = h;
    this->dt = dt;

    // Tools
    //this->dg = // TODO
    //this->db = // TODO
    this->t = 0;

    // Outputs
    this->title = "Rippling_Cuda";

    //print(dg, db);
    Device::assertDim(dg, db);
    }

Rippling::~Rippling()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/


/**
 * Override
 */
void Rippling::process(uchar4* ptrDevPixels, int w, int h)
    {
    // TODO lancer le kernel avec <<<dg,db>>>
    }


/**
 * Override
 */
void Rippling::animationStep()
    {
    // TODO
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float Rippling::getAnimationPara(void)
    {
    return t;
    }

/**
 * Override
 */
int Rippling::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int Rippling::getH(void)
    {
    return  h;
    }

/**
 * Override
 */
string Rippling::getTitle(void)
    {
    return title;
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

