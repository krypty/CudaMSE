#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "HeatTransfert.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void ecrasement(float* ptrImageInOutput, float* ptrImageHeater, int w, int h);
extern __global__ void diffusion(float* ptrImageInput, float* ptrImageOutput, int w, int h);
extern __global__ void toScreenImageHSB(uchar4* ptrDevPixels, float* ptrImageInput, int w, int h);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

HeatTransfert::HeatTransfert(int w, int h, float dt)
    {

    // Inputs
    this->w = w;
    this->h = h;
    this->dt = dt;

    // Tools
    this->dg = dim3(32, 4, 1);
    this->db = dim3(64, 8, 1);
    this->t = 0;
    this->isImageAInput = true;
    this->NB_ITERATION_AVEUGLE = 50;
    this->nbIterations = 0;

    // Outputs
    this->title = "HeatTransfert_Cuda";

    initImages();
    memoryManagment();

    //print(dg, db);
    Device::assertDim(dg, db);
    }

HeatTransfert::~HeatTransfert()
    {
    delete[] imageHeater;
    delete[] imageInit;

    HANDLE_ERROR(hipFree(ptrDevImageHeater));
    HANDLE_ERROR(hipFree(ptrDevImageInit));

    this->ptrDevImageInit = NULL;
    this->ptrDevImageHeater = NULL;
    this->ptrDevImageA = NULL;
    this->ptrDevImageB = NULL;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

void HeatTransfert::initImages()
    {
    imageHeater = new float[IMAGEWIDTH * IMAGEHEIGHT];

    for (int i = 0; i < IMAGEWIDTH; i++)
	{
	for (int j = 0; j < IMAGEHEIGHT; j++)
	    {

	    //Carré chaud au milieu
	    if (i > 350 && i < 450 && j > 350 && j < 450)
		imageHeater[i * IMAGEWIDTH + j] = 1.0;

	    //Petit carré froid en diagonale
	    if (i >= 280 && i <= 520 && (i <= 310 || i >= 490) && j >= 280 && j <= 520 && (j <= 310 || j >= 490))
		imageHeater[i * IMAGEWIDTH + j] = -0.2;

	    //Petit carré chaud
	    if (i >= 179 && i <= 621 && (i <= 195 || i >= 605 || (i >= 392 && i <= 408)) && j >= 179 && j <= 621 && (j <= 195 || j >= 605 || (j >= 392 && j <= 408)))
		imageHeater[i * IMAGEWIDTH + j] = 1.0;

	    }

	}

    imageInit = new float[IMAGEWIDTH * IMAGEHEIGHT];

    }

void HeatTransfert::memoryManagment()
    {
    //Global Memory

    this->ptrDevImageInit = NULL;
    this->ptrDevImageHeater = NULL;
    this->ptrDevImageA = NULL;
    this->ptrDevImageB = NULL;
    int size = IMAGEWIDTH * IMAGEHEIGHT * sizeof(float);

    HANDLE_ERROR(hipMalloc(&ptrDevImageInit, size));
    HANDLE_ERROR(hipMalloc(&ptrDevImageHeater, size));
    HANDLE_ERROR(hipMalloc(&ptrDevImageA, size));
    HANDLE_ERROR(hipMalloc(&ptrDevImageB, size));

    HANDLE_ERROR(hipMemset(ptrDevImageA, 0, size));
    HANDLE_ERROR(hipMemset(ptrDevImageB, 0, size));

    HANDLE_ERROR(hipMemcpy(ptrDevImageInit, this->imageInit, size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(ptrDevImageHeater, this->imageHeater, size, hipMemcpyHostToDevice));

    }

/**
 * Override
 */
void HeatTransfert::process(uchar4* ptrDevPixels, int w, int h)
    {

    float* ptrImageInput = NULL;
    float* ptrImageOutput = NULL;
    if (this->isImageAInput)
	{
	ptrImageInput = ptrDevImageA;
	ptrImageOutput = ptrDevImageB;
	}
    else
	{
	ptrImageInput = ptrDevImageB;
	ptrImageOutput = ptrDevImageA;
	}

    diffusion<<<dg,db>>>(ptrImageInput, ptrImageOutput, IMAGEWIDTH, IMAGEHEIGHT);

    ecrasement<<<dg,db>>>(ptrImageOutput, ptrDevImageHeater, IMAGEWIDTH, IMAGEHEIGHT);

    if(nbIterations % NB_ITERATION_AVEUGLE == 0)
	{
	    toScreenImageHSB<<<dg,db>>>(ptrDevPixels, ptrImageOutput, IMAGEWIDTH, IMAGEHEIGHT);
	}

    isImageAInput = !isImageAInput;
    nbIterations++;

    }

/**
 * Override
 */
void HeatTransfert::animationStep()
    {
    t += dt;
    }

/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
float HeatTransfert::getAnimationPara(void)
    {
    return t;
    }

/**
 * Override
 */
int HeatTransfert::getW(void)
    {
    return w;
    }

/**
 * Override
 */
int HeatTransfert::getH(void)
    {
    return h;
    }

/**
 * Override
 */
string HeatTransfert::getTitle(void)
    {
    return title;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
