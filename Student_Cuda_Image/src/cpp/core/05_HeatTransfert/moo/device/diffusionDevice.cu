#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void diffusion(float* ptrImageInput, float* ptrImageOutput, int w, int h);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float D_operator(float* ptrImage, int s, int w)
    {
    float k = 0.2;
    return ptrImage[s] + k * (ptrImage[s + w] + ptrImage[s - w] + ptrImage[s + 1] + ptrImage[s - 1] - 4 * ptrImage[s]);
    }

__global__ void diffusion(float* ptrImageInput, float* ptrImageOutput, int w, int h)
    {
    const int WH = w * h;

    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    int s = TID;

    while (s < WH)
	{
	// on n'applique pas la diffusion sur les bords
	if(s > w && s < WH-w && (w-1)%s != 0 && w%s != 0 )
	    ptrImageOutput[s] = D_operator(ptrImageInput, s, w);

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

