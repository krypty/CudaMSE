#include "hip/hip_runtime.h"
#include <iostream>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools.h"
#include "ColorTools.h"
#include "CalibreurF.h"
#include "IntervalF_GPU.h"

using std::cout;
using std::endl;
using gpu::IntervalF;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void toScreenImageHSB(uchar4* ptrDevPixels, float* ptrImageInput, int w, int h);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void toScreenImageHSB(uchar4* ptrDevPixels, float* ptrImageInput, int w, int h)
    {
    const int WH = w * h;

    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    int s = TID;

    float heatMax = 1.0;
    float heatMin = -0.2;
    float hueMax = 0;
    float hueMin = 0.66;

    CalibreurF calibreur(IntervalF(heatMin,heatMax),IntervalF(hueMin,hueMax));

    while (s < WH)
	{
	float hue = ptrImageInput[s];
	calibreur.calibrer(hue);
	uchar4 p;
	ColorTools::HSB_TO_RVB(hue,1,1,&p.x,&p.y,&p.z);
	ptrDevPixels[s] = p;
	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

