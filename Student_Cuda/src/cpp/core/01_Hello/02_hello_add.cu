#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include <stdio.h>
#include "cudaTools.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isAddScalarGPU_Ok(void);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ static int addScalarGPU(int a, int b);
__global__ static void addScalar(int a, int b, int* ptrC);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ bool isAddScalarGPU_Ok(void)
    {
    cout << endl << "[Hello Cuda 2]" << endl;

    int a = 2;
    int b = 7;

    int sumGPU = addScalarGPU(a, b);
    int sumTheorique = a + b;

    cout << "\n[CPU] " << a << " + " << b << " = " << sumGPU << endl;

    return sumGPU == sumTheorique;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__host__ int addScalarGPU(int a, int b)
    {
    int c;
    int* ptrC = &c;	// on host (CPU)
    int* ptrDev_c; 	// on device (GPU)

    // Specifier nb thread : ici 1 thread au total !
    dim3 dg = dim3(1, 1, 1);
    dim3 db = dim3(1, 1, 1);

    // Debug
    //Device::print(dg, db);
    Device::checkDimError(dg, db);

    size_t size = sizeof(int); // octet
    HANDLE_ERROR(hipMalloc((void** ) &ptrDev_c, size)); // Device memory allocation (*)

    addScalar<<<dg,db>>>(a,b,ptrDev_c); // asynchrone !!
    Device::checkLastCudaError("addScalar"); // facultatif

    //v1
    Device::synchronize(); // Pour printf sur  GPU

    //v2
    // hipDeviceSynchronize(); // Pour printf sur  GPU

    // memoryManagement => barrier de synchronisation
    HANDLE_ERROR(hipMemcpy(ptrC, ptrDev_c, size, hipMemcpyDeviceToHost));   // Device -> Host
    HANDLE_ERROR(hipFree(ptrDev_c)); // device dispose memory in (*)

    return c;
    }

/**
 * Hyp : 1 seul thread, ie dg(1,1,1) et db (1,1,1)
 */
__global__ void addScalar(int a, int b, int* ptrC)
    {
    *ptrC = a + b; // Hyp: 1 seul thread

    // debug
    printf("[GPU] %d + %d = %d", a, b, *ptrC);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

