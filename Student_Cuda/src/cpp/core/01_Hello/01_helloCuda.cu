#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include "cudaTools.h"
#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__  bool helloCuda(void); //__host__ facultatif

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ static void kernelHello(void);
__device__ static void doSomethingHello(void);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * resumer commande cuda:
 * http://developer.download.nvidia.com/compute/cuda/4_2/rel/toolkit/docs/online/index.html
 */
__host__ bool helloCuda(void) //__host__ facultatif
    {
    cout << endl << "[Hello Cuda 1]" << endl;

    // Specifier nb thread : ici 1 thread au total !
    dim3 dg = dim3(1, 1, 1);
    dim3 db = dim3(1, 1, 1);

    // Debug
    //Device::print(dg, db);
     Device::checkDimError(dg,db);

    kernelHello<<<dg,db>>>();  // asynchrone !!
    Device::checkLastCudaError("kernelHello"); // facultatif

    return true;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


/**
 * output : void
 */
__global__ void kernelHello(void)
    {
    doSomethingHello();
    }

/**
 * Can be call only by device
 * inliner by nvcc (nvidia compiler)
 */
__device__ void doSomethingHello(void)
    {
    // rien
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

