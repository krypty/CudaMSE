#include "hip/hip_runtime.h"
#include "ReductionTools.h"
#include <stdio.h>
#ifndef SRC_CPP_CORE_03_SAUCISSON_DEVICE_SAUCISSONDEVICE_CU_
#define SRC_CPP_CORE_03_SAUCISSON_DEVICE_SAUCISSONDEVICE_CU_


/*----------------------------------------------------------------------*\
|*			Declaration                     *|
\*---------------------------------------------------------------------*/

__global__ void saucissonDevice(float* ptrDevResult,int nbSaucisson,int n);
__device__ void initSM(float* tabSM,int n);
__device__ void peuplerSM(float* tabSM,int nbSaucisson);
__device__ float fpi(float x);

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

__global__ void saucissonDevice(float* ptrDevResult,int nbSaucisson,int n)
{
        extern __shared__ float tabSM[];
        ReductionTools<float,float> reductionTools = ReductionTools<float,float>(n);

        initSM(tabSM,n);
        __syncthreads();
        peuplerSM(tabSM,nbSaucisson);
        __syncthreads();
        reductionTools.reduction(tabSM,ptrDevResult);
}
__device__ void initSM(float* tabSM,int n)
{
        const int TID_LOCAL = Indice1D::tidLocal();
        const int NB_THREAD_LOCAL= Indice1D::nbThreadBlock();
        int s = TID_LOCAL;
        while(s < n)
        {
                tabSM[s]=0;
                s+=NB_THREAD_LOCAL;
        }
}
__device__ void peuplerSM(float* tabSM,int nbSaucisson)
{
        const int NB_THREAD=Indice1D::nbThread();
        const int TID = Indice1D::tid();
        const int TID_LOCAL = Indice1D::tidLocal();
        const float DX = 1.0f/(float)nbSaucisson;
        float s = TID;
        float sum = 0;

        while (s < nbSaucisson)
        {
                double xs = s * DX;
                sum += fpi(xs);
                s += NB_THREAD;
        }

        tabSM[TID_LOCAL] = sum;
}

__device__
float fpi(float x)
{
        return 4.0f / (1.0f + x * x);
}



#endif

/*----------------------------------------------------------------------*\
|*			End	                    *|
\*---------------------------------------------------------------------*/
