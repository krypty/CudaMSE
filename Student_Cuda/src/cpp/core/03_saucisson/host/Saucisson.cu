#include "hip/hip_runtime.h"
#include "Saucisson.h"
#include "ReductionTools.h"
#include <iostream>
#include <stdlib.h>
#include "Device.h"


using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration                     *|
\*---------------------------------------------------------------------*/

extern __global__ void saucissonDevice(float* ptrDevResult,int nbSaucisson,int n);

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/
Saucisson::Saucisson(int nbSaucisson)
{
        this->nbSaucisson = nbSaucisson;

        this->n = 1024;
        this->sizeSM = n*sizeof(float);
        memoryManagement();
        this->dg = dim3(64,1,1);
        this->db = dim3(1,1,1);

}
Saucisson::~Saucisson()
{
        HANDLE_ERROR(hipFree(ptrDevResult));
        ptrDevResult=NULL;
}

void Saucisson::process()
{
        saucissonDevice<<<dg,db,sizeSM>>>(ptrDevResult,nbSaucisson,n);
        Device::synchronize();
        HANDLE_ERROR(hipMemcpy(&pi,ptrDevResult,sizePI,hipMemcpyDeviceToHost));
}

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

void Saucisson::memoryManagement()
{
        ptrDevResult = NULL;
        sizePI = sizeof(float);
        HANDLE_ERROR(hipMalloc(&ptrDevResult,sizePI));

        HANDLE_ERROR(hipMemset(ptrDevResult,0,sizePI));
}
float Saucisson::getPi()
{
        return pi/nbSaucisson;
}

/*----------------------------------------------------------------------*\
|*			Implementation                  *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

/*----------------------------------------------------------------------*\
|*			End	                    *|
\*---------------------------------------------------------------------*/
