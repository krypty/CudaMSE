#include "hip/hip_runtime.h"
#include "MonteCarlo.h"
#include "ReductionTools.h"
#include <iostream>
#include <stdlib.h>
#include "Device.h"


using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration                     *|
\*---------------------------------------------------------------------*/

extern __global__ void monteCarloDevice(int* ptrDevResult,hiprandState* ptrDevTabGeneratorThread,int nbDarts,int n);
extern __global__ void setup_kernel_rand(hiprandState* ptrDevTabGeneratorThread, int deviceId);

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

MonteCarlo::MonteCarlo(int nbDarts)
{
    this->nbDarts = nbDarts;
    this->n = 1024;
    this->sizeSM = n * sizeof(int);
    this->sizePI = sizeof(int);
    this->dg = dim3(256,1,1);
    this->db = dim3(256,1,1);
    this->piInter = 0;
    this->pi = 0;
    this->nbDevices = 6;
    int nbThread = Device::nbThread(dg, db);

    ptrDevTabGeneratorThread = new hiprandState*[this->nbDevices];
    ptrDevResult = new int*[this->nbDevices];

    size_t generatorSize = sizeof(hiprandState) * nbThread;
    for(int i = 0; i < this->nbDevices; i++)
    {
	HANDLE_ERROR(hipSetDevice(i));
	HANDLE_ERROR(hipMalloc(&ptrDevResult[i], sizePI));

	HANDLE_ERROR(hipMalloc(&ptrDevTabGeneratorThread[i], generatorSize));
	setup_kernel_rand<<<dg, db>>>(ptrDevTabGeneratorThread[i], i);
    }
}
MonteCarlo::~MonteCarlo()
{
    HANDLE_ERROR(hipFree(*ptrDevResult));
    HANDLE_ERROR(hipFree(*ptrDevTabGeneratorThread));
    ptrDevResult=NULL;
    ptrDevTabGeneratorThread=NULL;
}

void MonteCarlo::process()
{
#pragma omp parallel for
    for(int i = 0; i < nbDevices; i++)
    {
	HANDLE_ERROR(hipSetDevice(i));
	HANDLE_ERROR(hipMemset(ptrDevResult[i], 0, sizePI));

	int tmp = 0;
	monteCarloDevice<<<dg, db, sizeSM>>>(ptrDevResult[i], ptrDevTabGeneratorThread[i], nbDarts / nbDevices, n);
	HANDLE_ERROR(hipMemcpy(&tmp, ptrDevResult[i], sizePI, hipMemcpyDeviceToHost));

//	cout << "result inter: " << tmp << endl;

#pragma omp atomic
	piInter += tmp;
    }
    pi = 4.0f * piInter /((float)nbDarts);
}

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

float MonteCarlo::getPi()
{
    return pi;
}

/*----------------------------------------------------------------------*\
|*			Implementation                  *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

/*----------------------------------------------------------------------*\
|*			End	                    *|
\*---------------------------------------------------------------------*/
