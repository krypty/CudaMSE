#include "hip/hip_runtime.h"
#include "MonteCarlo.h"
#include "ReductionTools.h"
#include <iostream>
#include <stdlib.h>
#include "Device.h"


using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration                     *|
\*---------------------------------------------------------------------*/

extern __global__ void monteCarloDevice(int* ptrDevResult,hiprandState* ptrDevTabGeneratorThread,int nbDarts,int n);
extern __global__ void setup_kernel_rand(hiprandState* ptrDevTabGeneratorThread, int deviceId);

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

MonteCarlo::MonteCarlo(int nbDarts)
{
        this->nbDarts = nbDarts;
        this->n = 1024;
        this->sizeSM = n*sizeof(int);
        memoryManagement();
        this->dg = dim3(64,1,1);
        this->db = dim3(1,1,1);
        this->piInter = 0;
        this->pi = 0;

        ptrDevTabGeneratorThread = new hiprandState*[Device::getDeviceCount()];
        ptrDevResult = new int*[Device::getDeviceCount()];

        size_t generatorSize = sizeof(hiprandState) * n;
        for(int i = 0; i < Device::getDeviceCount(); i++)
        {
                HANDLE_ERROR(hipSetDevice(i));
                HANDLE_ERROR(hipMalloc(&ptrDevResult[i], sizePI));
                HANDLE_ERROR(hipMemset(ptrDevResult[i], 0, sizePI));

                HANDLE_ERROR(hipMalloc(&ptrDevTabGeneratorThread[i], generatorSize));
                setup_kernel_rand<<<dg, db>>>(ptrDevTabGeneratorThread[i], Device::getDeviceId());
        }
}
MonteCarlo::~MonteCarlo()
{
        HANDLE_ERROR(hipFree(*ptrDevResult));
        HANDLE_ERROR(hipFree(*ptrDevTabGeneratorThread));
        ptrDevResult=NULL;
        ptrDevTabGeneratorThread=NULL;
}

void MonteCarlo::process()
{
	// chaque GPU va calculer pi et on renverra finalement la moyenne de chaque Pi calculé par tous les GPU
        int nbDevices = Device::getDeviceCount();

#pragma omp parallel for
        for(int i = 0; i < nbDevices; i++)
        {
                int tmp;
                HANDLE_ERROR(hipSetDevice(i));
                monteCarloDevice<<<dg, db, sizeSM>>>(ptrDevResult[i], ptrDevTabGeneratorThread[i], nbDarts / nbDevices, n);
                HANDLE_ERROR(hipMemcpy(&tmp, ptrDevResult[i], sizePI, hipMemcpyDeviceToHost));
    #pragma omp atomic
                piInter += tmp;
        }
        pi = 4.0f * piInter /((float)nbDarts);
}

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

void MonteCarlo::memoryManagement()
{
        ptrDevResult = NULL;
        sizePI = sizeof(float);
        HANDLE_ERROR(hipMalloc(&ptrDevResult, sizePI));

        HANDLE_ERROR(hipMemset(ptrDevResult, 0, sizePI));
}
float MonteCarlo::getPi()
{
        return pi;
}

/*----------------------------------------------------------------------*\
|*			Implementation                  *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

/*----------------------------------------------------------------------*\
|*			End	                    *|
\*---------------------------------------------------------------------*/
