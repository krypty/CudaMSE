#include "hip/hip_runtime.h"
#include "MonteCarlo.h"
#include "ReductionTools.h"
#include <iostream>
#include <stdlib.h>
#include "Device.h"


using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
|*			Declaration                     *|
\*---------------------------------------------------------------------*/

extern __global__ void monteCarloDevice(float* ptrDevResult,hiprandState* ptrDevTabGeneratorThread,int nbDarts,int n);
extern __global__ void setup_kernel_rand(hiprandState* ptrDevTabGeneratorThread, int deviceId);

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

MonteCarlo::MonteCarlo(int nbDarts)
{
        this->nbDarts = nbDarts;
        this->n = 1;
        this->sizeSM = n*sizeof(float);
        memoryManagement();
        this->dg = dim3(64,1,1);
        this->db = dim3(1,1,1);

        ptrDevTabGeneratorThread = new hiprandState*[Device::getDeviceCount()];
        ptrDevResult = new float*[Device::getDeviceCount()];

        size_t generatorSize = sizeof(hiprandState) * n;
        for(int i = 0; i < Device::getDeviceCount(); i++)
        {
                HANDLE_ERROR(hipSetDevice(i));
                HANDLE_ERROR(hipMalloc(&ptrDevResult[i], sizePI));
                HANDLE_ERROR(hipMemset(ptrDevResult[i], 0, sizePI));

                HANDLE_ERROR(hipMalloc(&ptrDevTabGeneratorThread[i], generatorSize));
                setup_kernel_rand<<<dg, db>>>(ptrDevTabGeneratorThread[i], Device::getDeviceId());
        }
}
MonteCarlo::~MonteCarlo()
{
        HANDLE_ERROR(hipFree(*ptrDevResult));
        HANDLE_ERROR(hipFree(*ptrDevTabGeneratorThread));
        ptrDevResult=NULL;
        ptrDevTabGeneratorThread=NULL;
}

void MonteCarlo::process()
{
        int nbDevices = Device::getDeviceCount();
#pragma omp parallel for
        for(int i = 0; i < nbDevices; i++)
        {
                float tmp;
                HANDLE_ERROR(hipSetDevice(i));
                monteCarloDevice<<<dg, db, sizeSM>>>(ptrDevResult[i], ptrDevTabGeneratorThread[i], nbDarts / nbDevices, n);
                HANDLE_ERROR(hipMemcpy(&tmp, ptrDevResult[i], sizePI, hipMemcpyDeviceToHost));
    #pragma omp atomic
                pi += tmp;
        }
        pi /= (float)nbDevices;
}

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

void MonteCarlo::memoryManagement()
{
        ptrDevResult = NULL;
        sizePI = sizeof(float);
        HANDLE_ERROR(hipMalloc(&ptrDevResult, sizePI));

        HANDLE_ERROR(hipMemset(ptrDevResult, 0, sizePI));
}
float MonteCarlo::getPi()
{
        return pi;
}

/*----------------------------------------------------------------------*\
|*			Implementation                  *|
\*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

/*--------------------------------------*\
|*		Private			*|
\*-------------------------------------*/

/*----------------------------------------------------------------------*\
|*			End	                    *|
\*---------------------------------------------------------------------*/
