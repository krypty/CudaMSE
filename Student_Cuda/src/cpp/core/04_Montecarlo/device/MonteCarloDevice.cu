#include "hip/hip_runtime.h"
#include "ReductionTools.h"
#include <stdio.h>
#include <limits.h>
#include "curandTools.h"
#include "Device.h"

#ifndef SRC_CPP_CORE_03_MONTECARLO_DEVICE_MONTECARLO_CU_
#define SRC_CPP_CORE_03_MONTECARLO_DEVICE_MONTECARLO_CU_


/*----------------------------------------------------------------------*\
|*			Declaration                     *|
\*---------------------------------------------------------------------*/

__global__ void monteCarloDevice(int* ptrDevResult, hiprandState* ptrDevTabGeneratorThread, int nbDarts, int n);
__device__ void initSharedMemory(int* tabSM, int n);
__device__ void peuplerSharedMemory(int* tabSM, hiprandState* ptrDevTabGeneratorThread, int nbDarts);

/*--------------------------------------*\
|*		Public			*|
\*-------------------------------------*/

__global__ void monteCarloDevice(int* ptrDevResult, hiprandState* ptrDevTabGeneratorThread,  int nbDarts, int n)
{
        extern __shared__ int tabSM[];

        ReductionTools<int, int> reductionTools = ReductionTools<int, int>(n);
        initSharedMemory(tabSM, n);
        __syncthreads();
        peuplerSharedMemory(tabSM, ptrDevTabGeneratorThread, nbDarts);
        __syncthreads();
        reductionTools.reduction(tabSM, ptrDevResult);
        __syncthreads();
        const int TID = Indice1D::tid();
}

__device__ void initSharedMemory(int* tabSM, int n)
{
        const int TID_LOCAL = Indice1D::tidLocal();
        const int NB_THREAD_LOCAL= Indice1D::nbThreadBlock();
        int s = TID_LOCAL;
        while(s < n)
        {
                tabSM[s] = 0;
                s += NB_THREAD_LOCAL;
        }
}
__device__ void peuplerSharedMemory(int* tabSM, hiprandState* ptrDevTabGeneratorThread, int nbDarts)
{
        const int NB_THREAD = Indice1D::nbThread();
        const int TID = Indice1D::tid();
        const int TID_LOCAL = Indice1D::tidLocal();
        float s = TID;

        int somme = 0;
        hiprandState localState = ptrDevTabGeneratorThread[TID];
        int tid = Indice1D::tid();

        while (s < nbDarts)
        {
                float x = hiprand_uniform(&localState);
                float y = hiprand_uniform(&localState);
                float z = 1.0f / (1.0f + x * x);
                if (y < z)
                {
                        somme += 1;
                }
                s += NB_THREAD;
        }
        tabSM[TID_LOCAL] = somme;
}

#endif

/*----------------------------------------------------------------------*\
|*			End	                    *|
\*---------------------------------------------------------------------*/
