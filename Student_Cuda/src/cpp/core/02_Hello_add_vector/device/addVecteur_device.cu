#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"

#include <stdio.h>



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void addVecteurGPU(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


static __device__ void secondaire(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void addVecteurGPU(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n)
    {
    secondaire(ptrDevV1, ptrDevV2, ptrDevW, n); // pas necessaire, just for fun
    }


/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


__device__ void secondaire(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n)
    {
    const int NB_THREAD=Indice2D::nbThread();
    const int TID=Indice2D::tid();

    // Debug
    if (TID==0)
	{
	printf("Coucou from device tid&d",TID); //required   Device::synchronize(); after the call of kernel
	}

   //TODO pattern entrelacement
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

