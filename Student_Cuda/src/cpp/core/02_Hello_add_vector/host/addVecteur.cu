#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include "Device.h"

#include <iostream>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addVecteurGPU(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);


/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__  void addVecteur(float* ptrV1, float* ptrV2, float* ptrW,int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/


__host__ void addVecteur(float* ptrV1, float* ptrV2, float* ptrW, int n)
    {
    float* ptrDevV1 = NULL;
    float* ptrDevV2 = NULL;
    float* ptrDevW = NULL;
    size_t size = n * sizeof(float); // octet

    HANDLE_ERROR(hipMalloc(&ptrDevV1, size));
    // TODO ptrV2 et ptrW

    HANDLE_ERROR(hipMemset(ptrDevW, 0, size));

    HANDLE_ERROR(hipMemcpy(ptrDevV1, ptrV1, size, hipMemcpyHostToDevice));
    // TODO ptrV2

    dim3 dg(16, 2, 1); // disons
    dim3 db(32, 4, 1); // disons

    // Debug
    //Device::print(dg, db);
    Device::checkDimError(dg, db);

    addVecteurGPU<<<dg,db>>>(ptrDevV1, ptrV2, ptrDevW, n); // assynchrone
    Device::checkLastCudaError("addVecteur"); // facultatif
    Device::synchronize(); // Temp, only for printf in  GPU

    HANDLE_ERROR(hipMemcpy(ptrW, ptrDevW, size, hipMemcpyDeviceToHost)); // barri�re synchronisation implicite

    HANDLE_ERROR(hipFree(ptrDevV1));
    // TODO ptrV2 et ptrW
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

